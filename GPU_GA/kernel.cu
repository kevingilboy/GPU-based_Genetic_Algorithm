/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <stdbool.h>

#include "params.h"
#include "simulator.c"
#include "ga.c"

/*
 * FUNCTION PROTOTYPES
 */
void pause();

int main()
{
	#ifdef CALC_PROGRAM_RUNTIME
		time_t start_time = time(NULL);
		clock_t timer;
	#endif

	//Stores the individuals of the population and their respective states
	//State is set to null if the protein is not used
	char population[INIT_POPULATION_SIZE][FULL_SIZE];

	//Stores the addr and errors of each individual resulting from simulation
	struct Individual individuals[INIT_POPULATION_SIZE];

	//Evolution counter
	int evolution = 0;

	srand(time(NULL));

	//Create the initial population of size INIT_POPULATION_SIZE
	DEBUG_PRINT(("E_%d: Initialization\n", evolution));
	START_TIMER
	initialize_population(population, INIT_POPULATION_SIZE);
	END_TIMER

	//Cycle through each individual
	DEBUG_PRINT(("E_%d: Simulation\n", evolution));
	START_TIMER
	for (int i = 0; i < INIT_POPULATION_SIZE; i++) {
		individuals[i].addr = population[i]; //Store the address
		individuals[i].error = simulate(population[i]); //Store the error
	}
	END_TIMER

	//Select the top SURVIVORS individuals
	DEBUG_PRINT(("E_%d: Natural Selection\n\n", evolution));
	START_TIMER
	natural_selection(individuals, INIT_POPULATION_SIZE);
	END_TIMER

	//At somepoint, if population becomes an array of mallocs, I should free population[MU:end]
	//However memory is not a major issue considering we only have a population of 300

	do {	
		//Increment evolution counter
		DEBUG_PRINT(("\n"));
		evolution++;

		//Duplicate SURVIVORS individuals to size TARGET_POPULATION_SIZE
		DEBUG_PRINT(("E_%d: Proliferation\n", evolution));
		START_TIMER
		proliferate(individuals);
		END_TIMER

		//Simulate each TARGET_POPULATION_SIZE individual
		DEBUG_PRINT(("E_%d: Simulation\n", evolution));
		START_TIMER
		for (int i = 0; i < TARGET_POPULATION_SIZE; i++) {
			individuals[i].error = simulate(individuals[i].addr);
		}
		END_TIMER

		//Select the top SURVIVORS individuals
		DEBUG_PRINT(("E_%d: Natural Selection\n", evolution));
		START_TIMER
		natural_selection(individuals, TARGET_POPULATION_SIZE);
		END_TIMER

		//Loop if any of the SURVIVORS individuals have nonzero error
	} while (!check_stopping_criteria(individuals));

	printf("Completed in %d evolutions\n", evolution);

	#ifdef CALC_PROGRAM_RUNTIME
		time_t end_time = time(NULL);
		printf("Runtime: %d sec\n\n", end_time - start_time);
	#endif

	#ifdef DEBUG
			int error = 0;
			for (int i = 0; i < SURVIVORS; i++) {
				for (int j = REDUCED_RULES; j < FULL_SIZE; j++) {
					if (individuals[i].addr[j] != -1) {
						printf("%s,", PROTEIN[j].name);
					}
				}
				printf("\n");

				error += individuals[i].error;
			}
			printf("E_%d: Total error of %d\n\n", evolution, error);
	#endif

	pause();
    return 0;
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}