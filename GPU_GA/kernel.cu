/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <stdbool.h>

#include "params.h"
#include "simulator.c"
#include "ga.c"

/*
 * FUNCTION PROTOTYPES
 */
void pause();

int main()
{
	//Stores the individuals of the population and their respective states
	//State is set to null if the protein is not used
	char population[INIT_POPULATION_SIZE][FULL_SIZE];

	//Stores the addr and errors of each individual resulting from simulation
	struct Individual individuals[INIT_POPULATION_SIZE];

	//Evolution counter
	int evolution = 0;

	srand(time(NULL));

	//Create the initial population of size INIT_POPULATION_SIZE
	initialize_population(population, INIT_POPULATION_SIZE);

	//Cycle through each individual
	for (int i = 0; i < INIT_POPULATION_SIZE; i++) {
		individuals[i].addr = population[i]; //Store the address
		individuals[i].error = simulate(population[i]); //Store the error
	}

	//Select the top SURVIVORS individuals
	natural_selection(individuals, INIT_POPULATION_SIZE);

	//At somepoint, if population becomes an array of mallocs, I should free population[MU:end]
	//However memory is not a major issue considering we only have a population of 300

	do {	
		//Duplicate SURVIVORS individuals to size TARGET_POPULATION_SIZE
		proliferate(individuals);

		//Simulate each TARGET_POPULATION_SIZE individual
		for (int i = 0; i < TARGET_POPULATION_SIZE; i++) {
			individuals[i].error = simulate(individuals[i].addr);
		}

		//Select the top SURVIVORS individuals
		natural_selection(individuals, TARGET_POPULATION_SIZE);

		//Increment evolution counter
		evolution++;

		//Loop if any of the SURVIVORS individuals have nonzero error
	} while (check_stopping_criteria(individuals));

	printf("Completed in %d evolutions\n", evolution);

	pause();
    return 0;
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}