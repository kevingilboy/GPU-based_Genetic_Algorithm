#include "hip/hip_runtime.h"

/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

#include "hip/hip_runtime.h"
#include ""

#include "params.h"
#include "simulator.c"
#include "tcell.c"

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

/*
* MACROS
*/
//Random number on range [min,max] inclusive
#define RANDGEN(min,max) (rand() % (max + 1 - min)) + min

/*
* FUNCTION PROTOTYPES
*/
void initialize_population();
bool check_stopping_criteria(int[], int);
void pause();

int main()
{
	//Stores the individuals of the population and their respective states
	//State is set to null if the protein is not used
	bool population[INIT_POPULATION_SIZE][FULL_SIZE];

	//Stores the errors of each individual resulting from simulation
	int error[INIT_POPULATION_SIZE];

	//The length of the population is stored separately so that the
	//population can be cropped without creating a new population array
	int population_size = INIT_POPULATION_SIZE;

	int evolution = 0;

	srand(time(NULL));

	initialize_population(population, population_size);

	do {
		for (int i = 0; i < population_size; i++) {
			error[i] = simulate(population[i]);

			//TODO sort population by increasing error

			//Trim to top 20 individuals
			population_size > 20 ? 20 : population_size;

			//TODO GA stuff here
		}

		//Increment evolution counter
		evolution++;
	} while (check_stopping_criteria(error, population_size));

	printf("Completed in %d evolutions\n", evolution);

	pause();
    return 0;
}

void initialize_population(bool population[INIT_POPULATION_SIZE][FULL_SIZE], int population_size) {
	//Cycle through each individual
	for (int i = 0; i < population_size; i++) {
		//Add the reduced rules in since those are
		//present in every individual
		for (int j = 0; j < REDUCED_SIZE; j++) {
			population[i][j] = PROTEINS[j].init_val;
		}
		for (int j = REDUCED_SIZE; j < FULL_SIZE; j++) {
			population[i][j] = NULL;
		}

		//Select TARGET_SIZE unique rules on the range
		//[REDUCED_SIZE,FULL_SIZE) and initialize them
		for (int j = 0; j < TARGET_SIZE; j++) {
			//Select a unique rule on range [REDUCED_SIZE,FULL_SIZE)
			int rule;
			do {
				rule = RANDGEN(REDUCED_SIZE,FULL_SIZE-1);
			} while (population[i][rule] != NULL);

			//Initialize the rule in the individual
			population[i][rule] = PROTEINS[rule].init_val;
		}
	}
}

bool check_stopping_criteria(int error[], int population_size) {
	for (int i = 0; i < population_size; i++) {
		//Return false on any nonzero error
		if (error[i] != 0) {
			return false;
		}
	}
	return true;
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}