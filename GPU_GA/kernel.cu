/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <stdbool.h>

#include "params.h"
#include "simulator.c"
#include "ga.c"

/*
 * FUNCTION PROTOTYPES
 */
void pause();

int main()
{
	//Stores the individuals of the population and their respective states
	//State is set to null if the protein is not used
	char population[INIT_POPULATION_SIZE][FULL_SIZE];

	//Stores the addr and errors of each individual resulting from simulation
	struct Individual individuals[INIT_POPULATION_SIZE];

	//Evolution counter
	int evolution = 0;

	srand(time(NULL));

	//Create the initial population of size INIT_POPULATION_SIZE
	DEBUG_PRINT(("E_%d: Initialization\n", evolution));
	initialize_population(population, INIT_POPULATION_SIZE);

	//Cycle through each individual
	DEBUG_PRINT(("E_%d: Simulation\n", evolution));
	for (int i = 0; i < INIT_POPULATION_SIZE; i++) {
		individuals[i].addr = population[i]; //Store the address
		individuals[i].error = simulate(population[i]); //Store the error
	}

	//Select the top SURVIVORS individuals
	DEBUG_PRINT(("E_%d: Natural Selection\n\n", evolution));
	natural_selection(individuals, INIT_POPULATION_SIZE);

	//At somepoint, if population becomes an array of mallocs, I should free population[MU:end]
	//However memory is not a major issue considering we only have a population of 300

	do {	
		//Increment evolution counter
		evolution++;

		//Duplicate SURVIVORS individuals to size TARGET_POPULATION_SIZE
		DEBUG_PRINT(("E_%d: Proliferation\n", evolution));
		proliferate(individuals);

		//Simulate each TARGET_POPULATION_SIZE individual
		DEBUG_PRINT(("E_%d: Simulation\n", evolution));
		for (int i = 0; i < TARGET_POPULATION_SIZE; i++) {
			individuals[i].error = simulate(individuals[i].addr);
		}

		//Select the top SURVIVORS individuals
		DEBUG_PRINT(("E_%d: Natural Selection\n", evolution));
		natural_selection(individuals, TARGET_POPULATION_SIZE);

		#ifdef DEBUG
				int error = 0;
				for (int i = 0; i < SURVIVORS; i++) {
					for (int j = REDUCED_RULES; j < FULL_SIZE; j++) {
						if (individuals[i].addr[j] != -1) {
							printf("%s,", PROTEIN[j].name);
						}
					}
					printf("\n");

					error += individuals[i].error;
				}
				printf("E_%d: Total error of %d\n\n", evolution, error);
		#endif

		//Loop if any of the SURVIVORS individuals have nonzero error
	} while (!check_stopping_criteria(individuals));

	printf("Completed in %d evolutions\n", evolution);

	pause();
    return 0;
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}