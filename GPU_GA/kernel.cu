/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include "params.h"
#include "simulator.c"
#include "ga.c"

/*
* FUNCTION PROTOTYPES
*/
void pause();

int main()
{
	//Stores the individuals of the population and their respective states
	//State is set to null if the protein is not used
	bool population[INIT_POPULATION_SIZE][FULL_SIZE];

	//Stores the errors of each individual resulting from simulation
	int error[INIT_POPULATION_SIZE];

	int evolution = 0;

	srand(time(NULL));

	initialize_population(population);

	//Cycle through each individual
	for (int i = 0; i < INIT_POPULATION_SIZE; i++) {
		error[i] = simulate(population[i]);
	}

	//Trim to top 20 individuals
	//TODO sort population by increasing error
	for (int keep_indices = 0; keep_indices < INDIVIDUALS_TO_KEEP; keep_indices++) {

	}

	do {
		//TODO GA stuff here

		//Cycle through each individual
		for (int i = 0; i < INDIVIDUALS_TO_KEEP; i++) {
			error[i] = simulate(population[i]);			
		}

		//Increment evolution counter
		evolution++;
	} while (check_stopping_criteria(error, INDIVIDUALS_TO_KEEP));

	printf("Completed in %d evolutions\n", evolution);

	pause();
    return 0;
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}