/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include "params.h"
#include "simulator.c"
#include "ga.c"

/*
* FUNCTION PROTOTYPES
*/
void pause();

int main()
{
	//Stores the individuals of the population and their respective states
	//State is set to null if the protein is not used
	bool population[INIT_POPULATION_SIZE][FULL_SIZE];

	//Stores the errors of each individual resulting from simulation
	int error[INIT_POPULATION_SIZE];

	int evolution = 0;

	srand(time(NULL));

	initialize_population(population);

	//Cycle through each individual
	for (int i = 0; i < INIT_POPULATION_SIZE; i++) {
		error[i] = simulate(population[i]);
	}

	//Trim to top 20 individuals
	//TODO sort population by increasing error
	for (int keep_indices = 0; keep_indices < SURVIVORS; keep_indices++) {

	}

	do {
		//TODO GA stuff here
		modify(population);

		//Cycle through each individual
		for (int i = 0; i < SURVIVORS; i++) {
			error[i] = simulate(population[i]);			
		}

		//Increment evolution counter
		evolution++;
	} while (check_stopping_criteria(error, SURVIVORS));

	printf("Completed in %d evolutions\n", evolution);

	pause();
    return 0;
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}