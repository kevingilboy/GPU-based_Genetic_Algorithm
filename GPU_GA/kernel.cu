
/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

#include "hip/hip_runtime.h"
#include ""
#include "simulator.c"
#include <stdio.h>

//Function declarations
void pause();

int main()
{
	printf("Hello world\n");

	pause();
    return 0;
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}