#include "hip/hip_runtime.h"

/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include "params.h"
#include "tcell.c"
#include "simulator.c"
#include "ga.c"

/*
* MACROS
*/
//Random number on range [min,max] inclusive
#define RANDGEN(min,max) (rand() % (max + 1 - min)) + min

/*
* FUNCTION PROTOTYPES
*/
void pause();

int main()
{
	//Stores the individuals of the population and their respective states
	//State is set to null if the protein is not used
	bool population[INIT_POPULATION_SIZE][FULL_SIZE];

	//Stores the errors of each individual resulting from simulation
	int error[INIT_POPULATION_SIZE];

	//The length of the population is stored separately so that the
	//population can be cropped without creating a new population array
	int population_size = INIT_POPULATION_SIZE;

	int evolution = 0;

	srand(time(NULL));

	initialize_population(population, population_size);

	do {
		for (int i = 0; i < population_size; i++) {
			error[i] = simulate(population[i]);

			//TODO sort population by increasing error

			//Trim to top 20 individuals
			population_size > 20 ? 20 : population_size;

			//TODO GA stuff here
		}

		//Increment evolution counter
		evolution++;
	} while (check_stopping_criteria(error, population_size));

	printf("Completed in %d evolutions\n", evolution);

	pause();
    return 0;
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}