#include "hip/hip_runtime.h"

/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

#include "hip/hip_runtime.h"
#include ""

#include "params.h"
#include "simulator.c"
#include "tcell.c"

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

//Random number on range [min,max] inclusive
#define RANDGEN(min,max) (rand() % (max + 1 - min)) + min

//Function declarations
void initialize_population();
void pause();

//Globals
bool population[INIT_POPULATION_SIZE][FULL_SIZE];
int error[INIT_POPULATION_SIZE];
int population_size = INIT_POPULATION_SIZE;

int main()
{
	srand(time(NULL));

	initialize_population();

	for (int i = 0; i < population_size; i++) {
		error[i] = simulate(population[i]);
	}

	pause();
    return 0;
}

void initialize_population() {
	//Cycle through each individual
	for (int i = 0; i < population_size; i++) {
		//Add the reduced rules in since those are
		//present in every individual
		for (int j = 0; j < REDUCED_SIZE; j++) {
			population[i][j] = PROTEINS[j].init_val;
		}
		for (int j = REDUCED_SIZE; j < FULL_SIZE; j++) {
			population[i][j] = NULL;
		}

		//Select TARGET_SIZE unique rules on the range
		//[REDUCED_SIZE,FULL_SIZE) and initialize them
		for (int j = 0; j < TARGET_SIZE; j++) {
			//Select a unique rule on range [REDUCED_SIZE,FULL_SIZE)
			int rule;
			do {
				rule = RANDGEN(REDUCED_SIZE,FULL_SIZE-1);
			} while (population[i][rule] != NULL);

			//Initialize the rule in the individual
			population[i][rule] = PROTEINS[rule].init_val;
		}
	}
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}