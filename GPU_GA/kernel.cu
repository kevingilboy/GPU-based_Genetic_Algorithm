/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include "params.h"
#include "simulator.c"
#include "ga.c"

/*
* FUNCTION PROTOTYPES
*/
void pause();

int main()
{
	//Stores the individuals of the population and their respective states
	//State is set to null if the protein is not used
	bool population[INIT_POPULATION_SIZE][FULL_SIZE];

	//Stores the addr and errors of each individual resulting from simulation
	struct Individual individuals[INIT_POPULATION_SIZE];

	//Evolution counter
	int evolution = 0;

	srand(time(NULL));

	//Create the initial population of size INIT_POPULATION_SIZE
	initialize_population((bool **)population, INIT_POPULATION_SIZE, FULL_SIZE, REDUCED_RULES, TARGET_RULES);

	//Cycle through each individual
	for (int i = 0; i < INIT_POPULATION_SIZE; i++) {
		individuals[i].addr = population[i]; //Store the address
		individuals[i].error = simulate(population[i]); //Store the error
	}

	//Select the top SURVIVORS individuals
	natural_selection(individuals, INIT_POPULATION_SIZE, FULL_SIZE, SURVIVORS);

	//At somepoint, if population becomes an array of mallocs, I should free population[MU:end]
	//However memory is not a major issue considering we only have a population of 300

	do {	
		//Duplicate SURVIVORS individuals to size TARGET_POPULATION_SIZE
		proliferate(individuals, SURVIVORS, FULL_SIZE, TARGET_POPULATION_SIZE);

		//Simulate each TARGET_POPULATION_SIZE individual
		for (int i = 0; i < TARGET_POPULATION_SIZE; i++) {
			individuals[i].error = simulate(population[i]);
		}

		//Select the top SURVIVORS individuals
		natural_selection(individuals, TARGET_POPULATION_SIZE, FULL_SIZE, SURVIVORS);

		//Increment evolution counter
		evolution++;

		//Loop if any of the SURVIVORS individuals have nonzero error
	} while (check_stopping_criteria(individuals, SURVIVORS));

	printf("Completed in %d evolutions\n", evolution);

	pause();
    return 0;
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}