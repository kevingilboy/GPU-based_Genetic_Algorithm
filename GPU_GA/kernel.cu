/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include "params.h"
#include "simulator.c"
#include "ga.c"

/*
* FUNCTION PROTOTYPES
*/
void pause();

int main()
{
	//Stores the individuals of the population and their respective states
	//State is set to null if the protein is not used
	bool population[INIT_POPULATION_SIZE][FULL_SIZE];

	//Stores the errors of each individual resulting from simulation
	int error[INIT_POPULATION_SIZE];

	int evolution = 0;

	srand(time(NULL));

	//Create the initial population of size INIT_POPULATION_SIZE
	initialize_population((bool **)population, INIT_POPULATION_SIZE, FULL_SIZE, REDUCED_RULES, TARGET_RULES);

	//Cycle through each individual
	for (int i = 0; i < INIT_POPULATION_SIZE; i++) {
		error[i] = simulate(population[i]);
	}

	//Select the top SURVIVORS individuals
	natural_selection((bool **)population, error, INIT_POPULATION_SIZE, FULL_SIZE, SURVIVORS);

	do {	
		//Duplicate SURVIVORS individuals to size TARGET_POPULATION_SIZE
		proliferate((bool **)population, SURVIVORS, FULL_SIZE, TARGET_POPULATION_SIZE);

		//Simulate each TARGET_POPULATION_SIZE individual
		for (int i = 0; i < TARGET_POPULATION_SIZE; i++) {
			error[i] = simulate(population[i]);			
		}

		//Select the top SURVIVORS individuals
		natural_selection((bool **)population, error, TARGET_POPULATION_SIZE, FULL_SIZE, SURVIVORS);

		//Increment evolution counter
		evolution++;

		//Loop if any of the SURVIVORS individuals have nonzero error
	} while (check_stopping_criteria(error, SURVIVORS));

	printf("Completed in %d evolutions\n", evolution);

	pause();
    return 0;
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}