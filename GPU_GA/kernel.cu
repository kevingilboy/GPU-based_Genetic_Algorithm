/*
 * Kevin Gilboy
 * GPU-ready main kernel
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <stdbool.h>

#include "params.h"
#include "ga.c"
#include "simulator.c"

/*
 * FUNCTION PROTOTYPES
 */
void pause();

int main()
{
	#ifdef CALC_PROGRAM_RUNTIME
		time_t start_time = time(NULL);
		clock_t timer;
	#endif

	//Stores the individuals of the population and their respective states
	//State is set to null if the protein is not used
	char population[INIT_POPULATION_SIZE][FULL_SIZE];

	//Stores the addr and errors of each individual resulting from simulation
	struct Individual individuals[INIT_POPULATION_SIZE];

	//Evolution counter
	int evolution = 0;

	srand(time(NULL));

	//Create the initial population of size INIT_POPULATION_SIZE
	DEBUG_PRINT(("E_%d: Initialization\n", evolution));
	START_TIMER
	initialize_population(population, INIT_POPULATION_SIZE);
	for (int i = 0; i < INIT_POPULATION_SIZE; i++) {
		individuals[i].addr = population[i]; //Store the address
	}
	END_TIMER

	//Cycle through each individual
	DEBUG_PRINT(("E_%d: Simulation\n", evolution));
	START_TIMER

#ifdef GPU
		hipError_t cudaStatus;
		char *gpu_individuals;
		float *cum_state;
		size_t pitch, pitch2;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			pause();
			exit(-1);
		}

		hipMallocPitch((void**) &gpu_individuals, &pitch, FULL_SIZE * sizeof(char), INIT_POPULATION_SIZE);
		hipMallocPitch((void**)&cum_state, &pitch2, FULL_SIZE * sizeof(float), INIT_POPULATION_SIZE);
		hipMemcpy2D(gpu_individuals, pitch, population, FULL_SIZE * sizeof(char), FULL_SIZE * sizeof(char), INIT_POPULATION_SIZE, hipMemcpyHostToDevice);

		int *gpu_error;
		hipMalloc((void**)&gpu_error, sizeof(int)*INIT_POPULATION_SIZE);

		int threads_per_block = 128;
		int blocks_per_grid = 300*200 / threads_per_block;
		simulate_parallel<<<blocks_per_grid, threads_per_block>>>(gpu_individuals, cum_state, pitch, pitch2, time(NULL)); //Store the error

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			pause();
			exit(-1);
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		//cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching simulate_parallel!\n", cudaStatus);
			pause();
			exit(-1);
		}

		//for (int r = 0; r < INIT_POPULATION_SIZE; ++r) {
			//int *row = (int*)((float*)cum_state + r * pitch2);
			float hostPtr[300][FULL_SIZE];
			//hipMemcpy(&e, row, sizeof(int) * INIT_POPULATION_SIZE, hipMemcpyDeviceToHost);
			hipMemcpy2D(hostPtr, FULL_SIZE * sizeof(float), cum_state, pitch, FULL_SIZE * sizeof(float), 300, hipMemcpyDeviceToHost);
			//printf("%d\n", e);
		//}
		printf("Done");
		pause();
		exit(0);
	#else
		for (int i = 0; i < INIT_POPULATION_SIZE; i++) {
			simulate(&individuals[i]); //Store the error
		}
	#endif
	END_TIMER

	//Select the top SURVIVORS individuals
	DEBUG_PRINT(("E_%d: Natural Selection\n\n", evolution));
	START_TIMER
	natural_selection(individuals, INIT_POPULATION_SIZE);
	END_TIMER

	//At somepoint, if population becomes an array of mallocs, I should free population[MU:end]
	//However memory is not a major issue considering we only have a population of 300

	do {	
		//Increment evolution counter
		DEBUG_PRINT(("\n"));
		evolution++;

		//Duplicate SURVIVORS individuals to size TARGET_POPULATION_SIZE
		DEBUG_PRINT(("E_%d: Proliferation\n", evolution));
		START_TIMER
		proliferate(individuals);
		END_TIMER

		//Simulate each TARGET_POPULATION_SIZE individual
		DEBUG_PRINT(("E_%d: Simulation\n", evolution));
		START_TIMER
		#ifdef GPU
			char *gpu_population;
			hipMalloc((void**)&gpu_population, sizeof(char) * 20 * FULL_SIZE);
			hipMemcpy(&gpu_population, &population, sizeof(char) * 20 * FULL_SIZE, hipMemcpyHostToDevice);

			Individual *gpu_individuals;
			hipMalloc((void**)&gpu_individuals, sizeof(Individual) * 20);
			for (int i = 0; i < 20; i++) {
				*gpu_individuals[i].addr = gpu_population[i];
			}

			//simulate_parallel<<<1, 1 >>>(&gpu_individuals, time(NULL)); //Store the error
		#else
			for (int i = 0; i < TARGET_POPULATION_SIZE; i++) {
				simulate(&individuals[i]); //Store the error
			}
		#endif
		END_TIMER

		//Select the top SURVIVORS individuals
		DEBUG_PRINT(("E_%d: Natural Selection\n", evolution));
		START_TIMER
		natural_selection(individuals, TARGET_POPULATION_SIZE);
		END_TIMER
#ifdef DEBUG
			int error = 0;
		for (int i = 0; i < SURVIVORS; i++) {
			for (int j = REDUCED_RULES; j < FULL_SIZE; j++) {
				if (individuals[i].addr[j] != -1) {
					printf("%s,", PROTEIN[j].name);
				}
			}
			printf("\n");

			error += individuals[i].error;
		}
		printf("E_%d: Total error of %d\n\n", evolution, error);
#endif
		//Loop if any of the SURVIVORS individuals have nonzero error
	} while (!check_stopping_criteria(individuals));

	printf("Completed in %d evolutions\n", evolution);

	#ifdef CALC_PROGRAM_RUNTIME
		time_t end_time = time(NULL);
		printf("Runtime: %d sec\n\n", end_time - start_time);
	#endif

	#ifdef DEBUG
			int error = 0;
			for (int i = 0; i < SURVIVORS; i++) {
				for (int j = REDUCED_RULES; j < FULL_SIZE; j++) {
					if (individuals[i].addr[j] != -1) {
						printf("%s,", PROTEIN[j].name);
					}
				}
				printf("\n");

				error += individuals[i].error;
			}
			printf("E_%d: Total error of %d\n\n", evolution, error);
	#endif

	pause();
    return 0;
}

void pause() {
	printf("Press [Enter] to continue . . .");
	fflush(stdout);
	getchar();
}